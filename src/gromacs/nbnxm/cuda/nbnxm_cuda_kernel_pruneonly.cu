#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2016,2017,2019,2020,2021, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
#include "gmxpre.h"

#include "nbnxm_cuda_kernel_pruneonly.cuh"

#include "gromacs/nbnxm/cuda/nbnxm_cuda.h"
#include "gromacs/nbnxm/gpu_types_common.h"
#include "gromacs/nbnxm/nbnxm_gpu_internal.h"

#ifndef FUNCTION_DECLARATION_ONLY
/* Instantiate external template functions */
template __global__ void
nbnxn_kernel_prune_cuda<false>(const NBAtomDataGpu, const NBParamGpu, const Nbnxm::gpu_plist, int, int);
template __global__ void
nbnxn_kernel_prune_cuda<true>(const NBAtomDataGpu, const NBParamGpu, const Nbnxm::gpu_plist, int, int);
#endif

namespace Nbnxm
{

/*! Calculates the amount of shared memory required by the CUDA kernel in use. */
static inline int calc_shmem_required_prune(const int num_threads_z)
{
    int shmem;

    /* i-atom x in shared memory */
    shmem = c_nbnxnGpuNumClusterPerSupercluster * c_clSize * sizeof(float4);
    /* cj in shared memory, for each warp separately */
    shmem += num_threads_z * c_nbnxnGpuClusterpairSplit * c_nbnxnGpuJgroupSize * sizeof(int);

    return shmem;
}

void launchNbnxmKernelPruneOnly(NbnxmGpu*                      nb,
                                const gmx::InteractionLocality iloc,
                                const int                      numParts,
                                const int                      part,
                                const int                      numSciInPart,
                                CommandEvent*                  timingEvent)
{
    NBAtomDataGpu*      adat         = nb->atdat;
    NBParamGpu*         nbp          = nb->nbparam;
    gpu_plist*          plist        = nb->plist[iloc];
    const DeviceStream& deviceStream = *nb->deviceStreams[iloc];

    /* Kernel launch config:
     * - The thread block dimensions match the size of i-clusters, j-clusters,
     *   and j-cluster concurrency, in x, y, and z, respectively.
     * - The 1D block-grid contains as many blocks as super-clusters.
     */
    int num_threads_z = c_pruneKernelJ4Concurrency;
    int nblock        = calc_nb_kernel_nblock(numSciInPart, &nb->deviceContext_->deviceInfo());
    KernelLaunchConfig config;
    config.blockSize[0]     = c_clSize;
    config.blockSize[1]     = c_clSize;
    config.blockSize[2]     = num_threads_z;
    config.gridSize[0]      = nblock;
    config.sharedMemorySize = calc_shmem_required_prune(num_threads_z);

    if (debug)
    {
        fprintf(debug,
                "Pruning GPU kernel launch configuration:\n\tThread block: %zux%zux%zu\n\t"
                "\tGrid: %zux%zu\n\t#Super-clusters/clusters: %d/%d (%d)\n"
                "\tShMem: %zu\n",
                config.blockSize[0],
                config.blockSize[1],
                config.blockSize[2],
                config.gridSize[0],
                config.gridSize[1],
                numSciInPart * c_nbnxnGpuNumClusterPerSupercluster,
                c_nbnxnGpuNumClusterPerSupercluster,
                plist->na_c,
                config.sharedMemorySize);
    }

    constexpr char kernelName[] = "k_pruneonly";
    const auto     kernel =
            plist->haveFreshList ? nbnxn_kernel_prune_cuda<true> : nbnxn_kernel_prune_cuda<false>;
    const auto kernelArgs = prepareGpuKernelArguments(kernel, config, adat, nbp, plist, &numParts, &part);
    launchGpuKernel(kernel, config, deviceStream, timingEvent, kernelName, kernelArgs);
}

} // namespace Nbnxm
