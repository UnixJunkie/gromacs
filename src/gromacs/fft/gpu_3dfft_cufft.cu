#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright 2016- The GROMACS Authors
 * and the project initiators Erik Lindahl, Berk Hess and David van der Spoel.
 * Consult the AUTHORS/COPYING files and https://www.gromacs.org for details.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * https://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at https://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out https://www.gromacs.org.
 */

/*! \internal \file
 *  \brief Implements GPU 3D FFT routines for CUDA.
 *
 *  \author Aleksei Iupinov <a.yupinov@gmail.com>
 *  \author Mark Abraham <mark.j.abraham@gmail.com>
 *  \ingroup module_fft
 */

#include "gmxpre.h"
#include <hipfft/hipfftXt.h>
#include "gpu_3dfft_cufft.h"

#include "gromacs/gpu_utils/device_stream.h"
#include "gromacs/gpu_utils/devicebuffer.h"
#include "gromacs/utility/arrayref.h"
#include "gromacs/utility/fatalerror.h"
#include "gromacs/utility/gmxassert.h"

#include <iostream>

namespace gmx
{
static void handleCufftError(hipfftResult_t status, const char* msg)
{
    if (status != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "%s (error code %d)\n", msg, status);
    }
}

Gpu3dFft::ImplCuFft::ImplCuFft(bool allocateRealGrid,
                               MPI_Comm /*comm*/,
                               ArrayRef<const int> gridSizesInXForEachRank,
                               ArrayRef<const int> gridSizesInYForEachRank,
                               const int /*nz*/,
                               bool                 performOutOfPlaceFFT,
                               const DeviceContext& context,
                               const DeviceStream&  pmeStream,
                               ivec                 realGridSize,
                               ivec                 realGridSizePadded,
                               ivec                 complexGridSizePadded,
                               DeviceBuffer<__half>* realGrid,
                               DeviceBuffer<__half>* complexGrid) :
    Gpu3dFft::Impl::Impl(performOutOfPlaceFFT)
{
    GMX_RELEASE_ASSERT(allocateRealGrid == true, "Grids cannot be pre-allocated");
    GMX_RELEASE_ASSERT(gridSizesInXForEachRank.size() == 1 && gridSizesInYForEachRank.size() == 1,
                       "FFT decomposition not implemented with cuFFT backend");

    const int complexGridSizePaddedTotal =
            complexGridSizePadded[XX] * complexGridSizePadded[YY] * complexGridSizePadded[ZZ];
    const int realGridSizePaddedTotal =
            realGridSizePadded[XX] * realGridSizePadded[YY] * realGridSizePadded[ZZ];

    allocateDeviceBuffer(realGrid, realGridSizePaddedTotal, context);
    allocateComplexGrid(complexGridSizePadded, realGrid, complexGrid, context);

    realGrid_ = reinterpret_cast<__half*>(*realGrid);

    GMX_RELEASE_ASSERT(realGrid_, "Bad (null) input real-space grid");
    GMX_RELEASE_ASSERT(complexGrid_, "Bad (null) input complex grid");

    std::cout << "FFT grid " << realGridSize[XX] << " " << realGridSize[YY] << " " << realGridSize[ZZ] << std::endl;

    hipfftResult_t result;
    /* Commented code for a simple 3D grid with no padding */
    /*
       result = hipfftPlan3d(&planR2C_, realGridSize[XX], realGridSize[YY], realGridSize[ZZ],
       HIPFFT_R2C); handleCufftError(result, "hipfftPlan3d R2C plan failure");

       result = hipfftPlan3d(&planC2R_, realGridSize[XX], realGridSize[YY], realGridSize[ZZ],
       HIPFFT_C2R); handleCufftError(result, "hipfftPlan3d C2R plan failure");
     */

    const int rank = 3, batch = 1;
    size_t workSize = 0;
    result = hipfftCreate(&planR2C_);
    handleCufftError(result, "hipfftCreate failure");

    long long int rgsize[DIM];
    rgsize[0] = realGridSize[0];
    rgsize[1] = realGridSize[1];
    rgsize[2] = realGridSize[2];

    long long int rgsizepad[DIM];
    rgsizepad[0] = realGridSizePadded[0];
    rgsizepad[1] = realGridSizePadded[1];
    rgsizepad[2] = realGridSizePadded[2];

    long long int cgsizepad[DIM];
    cgsizepad[0] = complexGridSizePadded[0];
    cgsizepad[1] = complexGridSizePadded[1];
    cgsizepad[2] = complexGridSizePadded[2];

    result = hipfftXtMakePlanMany(planR2C_, rank, rgsize, rgsizepad, 1, realGridSizePaddedTotal,
                                HIP_R_16F, cgsizepad, 1, complexGridSizePaddedTotal, HIP_C_16F, batch, 
                                &workSize, HIP_C_16F);
    handleCufftError(result, "hipfftXtMakePlanMany R2C plan failure");

    // result = hipfftPlanMany(&planR2C_,
    //                        rank,
    //                        realGridSize,
    //                        realGridSizePadded,
    //                        1,
    //                        realGridSizePaddedTotal,
    //                        complexGridSizePadded,
    //                        1,
    //                        complexGridSizePaddedTotal,
    //                        HIPFFT_R2C,
    //                        batch);
    // handleCufftError(result, "hipfftPlanMany R2C plan failure");

    result = hipfftCreate(&planC2R_);
    handleCufftError(result, "hipfftCreate failure");

    result = hipfftXtMakePlanMany(planC2R_, rank, rgsize, cgsizepad, 1, complexGridSizePaddedTotal,
                                HIP_C_16F, rgsizepad, 1, realGridSizePaddedTotal, HIP_R_16F, batch, 
                                &workSize, HIP_R_16F);
    handleCufftError(result, "hipfftXtMakePlanMany C2R plan failure");

    // result = hipfftPlanMany(&planC2R_,
    //                        rank,
    //                        realGridSize,
    //                        complexGridSizePadded,
    //                        1,
    //                        complexGridSizePaddedTotal,
    //                        realGridSizePadded,
    //                        1,
    //                        realGridSizePaddedTotal,
    //                        HIPFFT_C2R,
    //                        batch);
    // handleCufftError(result, "hipfftPlanMany C2R plan failure");

    hipStream_t stream = pmeStream.stream();
    GMX_RELEASE_ASSERT(stream, "Can not use the default CUDA stream for PME cuFFT");

    result = hipfftSetStream(planR2C_, stream);
    handleCufftError(result, "hipfftSetStream R2C failure");

    result = hipfftSetStream(planC2R_, stream);
    handleCufftError(result, "hipfftSetStream C2R failure");
}

Gpu3dFft::ImplCuFft::~ImplCuFft()
{
    deallocateComplexGrid();

    hipfftResult_t result;
    result = hipfftDestroy(planR2C_);
    handleCufftError(result, "hipfftDestroy R2C failure");
    result = hipfftDestroy(planC2R_);
    handleCufftError(result, "hipfftDestroy C2R failure");
}

void Gpu3dFft::ImplCuFft::perform3dFft(gmx_fft_direction dir, CommandEvent* /*timingEvent*/)
{
    hipfftResult_t result;
    if (dir == GMX_FFT_REAL_TO_COMPLEX)
    {
        result = hipfftXtExec(planR2C_, realGrid_, complexGrid_, HIPFFT_FORWARD);
        handleCufftError(result, "cuFFT R2C execution failure");
    }
    else
    {
        result = hipfftXtExec(planC2R_, complexGrid_, realGrid_, HIPFFT_BACKWARD);
        handleCufftError(result, "cuFFT C2R execution failure");
    }
}

} // namespace gmx
