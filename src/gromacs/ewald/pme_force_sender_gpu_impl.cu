/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2019,2020, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements PME-PP communication using CUDA
 *
 *
 * \author Alan Gray <alang@nvidia.com>
 *
 * \ingroup module_ewald
 */
#include "gmxpre.h"

#include "pme_force_sender_gpu_impl.h"

#include "config.h"

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/gpueventsynchronizer.cuh"
#include "gromacs/utility/gmxmpi.h"

namespace gmx
{

/*! \brief Create PME-PP GPU communication object */
PmeForceSenderGpu::Impl::Impl(const DeviceStream& pmeStream, MPI_Comm comm, gmx::ArrayRef<PpRanks> ppRanks) :
    pmeStream_(pmeStream),
    comm_(comm),
    ppRanks_(ppRanks)
{
    request_.resize(ppRanks.size(), MPI_REQUEST_NULL);
}

PmeForceSenderGpu::Impl::~Impl()
{
#if GMX_MPI

#    if GMX_THREAD_MPI
#    else
    // free resources as MPI_waitall might not get called on these requests
    std::for_each(request_.begin(), request_.end(), [](MPI_Request& req) {
        if (req != MPI_REQUEST_NULL)
        {
            MPI_Request_free(&req);
            req = MPI_REQUEST_NULL;
        }
    });
#    endif
#endif
}

/*! \brief  sends force buffer address to PP ranks */
void PmeForceSenderGpu::Impl::setForceBufferAddress(rvec* d_f)
{
#if GMX_MPI

#    if GMX_THREAD_MPI
    int ind_start = 0;
    int ind_end   = 0;
    for (const auto& receiver : ppRanks_)
    {
        ind_start = ind_end;
        ind_end   = ind_start + receiver.numAtoms;

        // Data will be transferred directly from GPU.
        void* sendBuf = reinterpret_cast<void*>(&d_f[ind_start]);

        MPI_Send(&sendBuf, sizeof(void**), MPI_BYTE, receiver.rankId, 0, comm_);
    }
#    else
    // Just store the pointer which will be sent later
    d_f_ = d_f;
#    endif

#else
    GMX_UNUSED_VALUE(d_f);
#endif
}

/*! \brief Send PME data directly using CUDA memory copy */
void PmeForceSenderGpu::Impl::sendFToPp(int ppRank)
{
#if GMX_MPI
#    if GMX_THREAD_MPI
    // Data will be pulled directly from PP task

    // Record and send event to ensure PME force calcs are completed before PP task pulls data
    pmeSync_.markEvent(pmeStream_);
    GpuEventSynchronizer* pmeSyncPtr = &pmeSync_;

    // TODO Using MPI_Isend would be more efficient, particularly when
    // sending to multiple PP ranks
    MPI_Send(&pmeSyncPtr, sizeof(GpuEventSynchronizer*), MPI_BYTE, ppRank, 0, comm_);

#    else  // ToDo: split the logic in different functions

    int ind_start = 0;
    int i         = 0;
    // Calculate starting atom for given PP rank
    for (; i < ppRanks_.size() && ppRanks_[i].rankId != ppRank; ++i)
    {
        ind_start += ppRanks_[i].numAtoms;
    }

    GMX_ASSERT(i < ppRanks_.size(), "ppRank value different from expected values");

    // This is needed to free resources; MPI_ISend call below is expected to be finished by now as
    // PP rank has MPI_Wait to receive the data.
    if (request_[i] != MPI_REQUEST_NULL)
    {
        MPI_Request_free(&request_[i]);
        request_[i] = MPI_REQUEST_NULL;
    }

    // Ensure PME force calcs are completed before data is sent
    hipError_t stat = hipStreamSynchronize(pmeStream_.stream());
    CU_RET_ERR(stat, "hipStreamSynchronize on pmeStream_ failed");

    MPI_Isend(&d_f_[ind_start], ppRanks_[i].numAtoms * sizeof(rvec), MPI_BYTE, ppRank, 0, comm_,
              &request_[i]);
#    endif // GMX_THREAD_MPI
#else
    GMX_UNUSED_VALUE(pmeSyncPtr);
    GMX_UNUSED_VALUE(ppRank);
#endif
}

PmeForceSenderGpu::PmeForceSenderGpu(const DeviceStream&    pmeStream,
                                     MPI_Comm               comm,
                                     gmx::ArrayRef<PpRanks> ppRanks) :
    impl_(new Impl(pmeStream, comm, ppRanks))
{
}

PmeForceSenderGpu::~PmeForceSenderGpu() = default;

void PmeForceSenderGpu::setForceBufferAddress(rvec* d_f)
{
    impl_->setForceBufferAddress(d_f);
}

void PmeForceSenderGpu::sendFToPp(int ppRank)
{
    impl_->sendFToPp(ppRank);
}


} // namespace gmx
