#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2018,2019,2020,2021, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements CUDA bonded functionality
 *
 * \author Jon Vincent <jvincent@nvidia.com>
 * \author Magnus Lundborg <lundborg.magnus@gmail.com>
 * \author Berk Hess <hess@kth.se>
 * \author Szilárd Páll <pall.szilard@gmail.com>
 * \author Alan Gray <alang@nvidia.com>
 * \author Mark Abraham <mark.j.abraham@gmail.com>
 *
 * \ingroup module_listed_forces
 */

#include "gmxpre.h"

#include <cassert>

#include <hip/hip_math_constants.h>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/typecasts.cuh"
#include "gromacs/gpu_utils/vectype_ops.cuh"
#include "gromacs/listed_forces/listed_forces_gpu.h"
#include "gromacs/math/units.h"
#include "gromacs/mdlib/force_flags.h"
#include "gromacs/mdtypes/interaction_const.h"
#include "gromacs/mdtypes/simulation_workload.h"
#include "gromacs/pbcutil/pbc_aiuc_cuda.cuh"
#include "gromacs/timing/wallcycle.h"
#include "gromacs/utility/gmxassert.h"

#include "listed_forces_gpu_impl.h"

#if defined(_MSVC)
#    include <limits>
#endif

/*-------------------------------- CUDA kernels-------------------------------- */
/*------------------------------------------------------------------------------*/

#define CUDA_DEG2RAD_F (HIP_PI_F / 180.0F)
#define CUDA_RAD2DEG_F (180.0F / HIP_PI_F)

namespace
{
/*! \brief Mysterious CMAP coefficient matrix */
__device__ const int cmap_coeff_matrix[] = {
    1,  0,  -3, 2,  0,  0, 0,  0,  -3, 0,  9,  -6, 2, 0,  -6, 4,  0,  0,  0, 0,  0, 0, 0,  0,
    3,  0,  -9, 6,  -2, 0, 6,  -4, 0,  0,  0,  0,  0, 0,  0,  0,  0,  0,  9, -6, 0, 0, -6, 4,
    0,  0,  3,  -2, 0,  0, 0,  0,  0,  0,  -9, 6,  0, 0,  6,  -4, 0,  0,  0, 0,  1, 0, -3, 2,
    -2, 0,  6,  -4, 1,  0, -3, 2,  0,  0,  0,  0,  0, 0,  0,  0,  -1, 0,  3, -2, 1, 0, -3, 2,
    0,  0,  0,  0,  0,  0, 0,  0,  0,  0,  -3, 2,  0, 0,  3,  -2, 0,  0,  0, 0,  0, 0, 3,  -2,
    0,  0,  -6, 4,  0,  0, 3,  -2, 0,  1,  -2, 1,  0, 0,  0,  0,  0,  -3, 6, -3, 0, 2, -4, 2,
    0,  0,  0,  0,  0,  0, 0,  0,  0,  3,  -6, 3,  0, -2, 4,  -2, 0,  0,  0, 0,  0, 0, 0,  0,
    0,  0,  -3, 3,  0,  0, 2,  -2, 0,  0,  -1, 1,  0, 0,  0,  0,  0,  0,  3, -3, 0, 0, -2, 2,
    0,  0,  0,  0,  0,  1, -2, 1,  0,  -2, 4,  -2, 0, 1,  -2, 1,  0,  0,  0, 0,  0, 0, 0,  0,
    0,  -1, 2,  -1, 0,  1, -2, 1,  0,  0,  0,  0,  0, 0,  0,  0,  0,  0,  1, -1, 0, 0, -1, 1,
    0,  0,  0,  0,  0,  0, -1, 1,  0,  0,  2,  -2, 0, 0,  -1, 1
};

} // namespace

/*---------------- BONDED CUDA kernels--------------*/

/* Harmonic */
__device__ __forceinline__ static void
harmonic_gpu(const float kA, const float xA, const float x, float* V, float* F)
{
    constexpr float half = 0.5F;
    float           dx, dx2;

    dx  = x - xA;
    dx2 = dx * dx;

    *F = -kA * dx;
    *V = half * kA * dx2;
}

template<bool calcVir, bool calcEner>
__device__ void bonds_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          const float4    gm_xq[],
                          float3          gm_f[],
                          float3          sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        const int3 bondData = *(reinterpret_cast<const int3*>(d_forceatoms + 3 * i));
        int        type     = bondData.x;
        int        ai       = bondData.y;
        int        aj       = bondData.z;

        /* dx = xi - xj, corrected for periodic boundary conditions. */
        float3 dx;
        int    ki = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[aj], dx);

        float dr2 = norm2(dx);
        float dr  = sqrt(dr2);

        float vbond;
        float fbond;
        harmonic_gpu(d_forceparams[type].harmonic.krA, d_forceparams[type].harmonic.rA, dr, &vbond, &fbond);

        if (calcEner)
        {
            *vtot_loc += vbond;
        }

        if (dr2 != 0.0F)
        {
            fbond *= rsqrtf(dr2);

            float3 fij = fbond * dx;
            atomicAdd(&gm_f[ai], fij);
            atomicAdd(&gm_f[aj], -fij);
            if (calcVir && ki != gmx::c_centralShiftIndex)
            {
                atomicAdd(&sm_fShiftLoc[ki], fij);
                atomicAdd(&sm_fShiftLoc[gmx::c_centralShiftIndex], -fij);
            }
        }
    }
}

template<bool returnShift>
__device__ __forceinline__ static float bond_angle_gpu(const float4   xi,
                                                       const float4   xj,
                                                       const float4   xk,
                                                       const PbcAiuc& pbcAiuc,
                                                       float3*        r_ij,
                                                       float3*        r_kj,
                                                       float*         costh,
                                                       int*           t1,
                                                       int*           t2)
/* Return value is the angle between the bonds i-j and j-k */
{
    *t1 = pbcDxAiuc<returnShift>(pbcAiuc, xi, xj, *r_ij);
    *t2 = pbcDxAiuc<returnShift>(pbcAiuc, xk, xj, *r_kj);

    *costh   = cos_angle(*r_ij, *r_kj);
    float th = acosf(*costh);

    return th;
}

template<bool calcVir, bool calcEner>
__device__ void angles_gpu(const int       i,
                           float*          vtot_loc,
                           const int       numBonds,
                           const t_iatom   d_forceatoms[],
                           const t_iparams d_forceparams[],
                           const float4    gm_xq[],
                           float3          gm_f[],
                           float3          sm_fShiftLoc[],
                           const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        const int4 angleData = *(reinterpret_cast<const int4*>(d_forceatoms + 4 * i));
        int        type      = angleData.x;
        int        ai        = angleData.y;
        int        aj        = angleData.z;
        int        ak        = angleData.w;

        float3 r_ij;
        float3 r_kj;
        float  cos_theta;
        int    t1;
        int    t2;
        float  theta = bond_angle_gpu<calcVir>(
                gm_xq[ai], gm_xq[aj], gm_xq[ak], pbcAiuc, &r_ij, &r_kj, &cos_theta, &t1, &t2);

        float va;
        float dVdt;
        harmonic_gpu(d_forceparams[type].harmonic.krA,
                     d_forceparams[type].harmonic.rA * CUDA_DEG2RAD_F,
                     theta,
                     &va,
                     &dVdt);

        if (calcEner)
        {
            *vtot_loc += va;
        }

        float cos_theta2 = cos_theta * cos_theta;
        if (cos_theta2 < 1.0F)
        {
            float st    = dVdt * rsqrtf(1.0F - cos_theta2);
            float sth   = st * cos_theta;
            float nrij2 = norm2(r_ij);
            float nrkj2 = norm2(r_kj);

            float nrij_1 = rsqrtf(nrij2);
            float nrkj_1 = rsqrtf(nrkj2);

            float cik = st * nrij_1 * nrkj_1;
            float cii = sth * nrij_1 * nrij_1;
            float ckk = sth * nrkj_1 * nrkj_1;

            float3 f_i = cii * r_ij - cik * r_kj;
            float3 f_k = ckk * r_kj - cik * r_ij;
            float3 f_j = -f_i - f_k;

            atomicAdd(&gm_f[ai], f_i);
            atomicAdd(&gm_f[aj], f_j);
            atomicAdd(&gm_f[ak], f_k);

            if (calcVir)
            {
                atomicAdd(&sm_fShiftLoc[t1], f_i);
                atomicAdd(&sm_fShiftLoc[gmx::c_centralShiftIndex], f_j);
                atomicAdd(&sm_fShiftLoc[t2], f_k);
            }
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void urey_bradley_gpu(const int       i,
                                 float*          vtot_loc,
                                 const int       numBonds,
                                 const t_iatom   d_forceatoms[],
                                 const t_iparams d_forceparams[],
                                 const float4    gm_xq[],
                                 float3          gm_f[],
                                 float3          sm_fShiftLoc[],
                                 const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        const int4 ubData = *(reinterpret_cast<const int4*>(d_forceatoms + 4 * i));
        int        type   = ubData.x;
        int        ai     = ubData.y;
        int        aj     = ubData.z;
        int        ak     = ubData.w;

        float th0A = d_forceparams[type].u_b.thetaA * CUDA_DEG2RAD_F;
        float kthA = d_forceparams[type].u_b.kthetaA;
        float r13A = d_forceparams[type].u_b.r13A;
        float kUBA = d_forceparams[type].u_b.kUBA;

        float3 r_ij;
        float3 r_kj;
        float  cos_theta;
        int    t1;
        int    t2;
        float  theta = bond_angle_gpu<calcVir>(
                gm_xq[ai], gm_xq[aj], gm_xq[ak], pbcAiuc, &r_ij, &r_kj, &cos_theta, &t1, &t2);

        float va;
        float dVdt;
        harmonic_gpu(kthA, th0A, theta, &va, &dVdt);

        if (calcEner)
        {
            *vtot_loc += va;
        }

        float3 r_ik;
        int    ki = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[ak], r_ik);

        float dr2 = norm2(r_ik);
        float dr  = dr2 * rsqrtf(dr2);

        float vbond;
        float fbond;
        harmonic_gpu(kUBA, r13A, dr, &vbond, &fbond);

        float cos_theta2 = cos_theta * cos_theta;
        if (cos_theta2 < 1.0F)
        {
            float st  = dVdt * rsqrtf(1.0F - cos_theta2);
            float sth = st * cos_theta;

            float nrkj2 = norm2(r_kj);
            float nrij2 = norm2(r_ij);

            float cik = st * rsqrtf(nrkj2 * nrij2);
            float cii = sth / nrij2;
            float ckk = sth / nrkj2;

            float3 f_i = cii * r_ij - cik * r_kj;
            float3 f_k = ckk * r_kj - cik * r_ij;
            float3 f_j = -f_i - f_k;

            atomicAdd(&gm_f[ai], f_i);
            atomicAdd(&gm_f[aj], f_j);
            atomicAdd(&gm_f[ak], f_k);

            if (calcVir)
            {
                atomicAdd(&sm_fShiftLoc[t1], f_i);
                atomicAdd(&sm_fShiftLoc[gmx::c_centralShiftIndex], f_j);
                atomicAdd(&sm_fShiftLoc[t2], f_k);
            }
        }

        /* Time for the bond calculations */
        if (dr2 != 0.0F)
        {
            if (calcEner)
            {
                *vtot_loc += vbond;
            }

            fbond *= rsqrtf(dr2);

            float3 fik = fbond * r_ik;
            atomicAdd(&gm_f[ai], fik);
            atomicAdd(&gm_f[ak], -fik);

            if (calcVir && ki != gmx::c_centralShiftIndex)
            {
                atomicAdd(&sm_fShiftLoc[ki], fik);
                atomicAdd(&sm_fShiftLoc[gmx::c_centralShiftIndex], -fik);
            }
        }
    }
}

template<bool returnShift, typename T>
__device__ __forceinline__ static float dih_angle_gpu(const T        xi,
                                                      const T        xj,
                                                      const T        xk,
                                                      const T        xl,
                                                      const PbcAiuc& pbcAiuc,
                                                      float3*        r_ij,
                                                      float3*        r_kj,
                                                      float3*        r_kl,
                                                      float3*        m,
                                                      float3*        n,
                                                      int*           t1,
                                                      int*           t2,
                                                      int*           t3)
{
    *t1 = pbcDxAiuc<returnShift>(pbcAiuc, xi, xj, *r_ij);
    *t2 = pbcDxAiuc<returnShift>(pbcAiuc, xk, xj, *r_kj);
    *t3 = pbcDxAiuc<returnShift>(pbcAiuc, xk, xl, *r_kl);

    *m         = cprod(*r_ij, *r_kj);
    *n         = cprod(*r_kj, *r_kl);
    float phi  = gmx_angle(*m, *n);
    float ipr  = iprod(*r_ij, *n);
    float sign = (ipr < 0.0F) ? -1.0F : 1.0F;
    phi        = sign * phi;

    return phi;
}


__device__ __forceinline__ static void
dopdihs_gpu(const float cpA, const float phiA, const int mult, const float phi, float* v, float* f)
{
    float mdphi, sdphi;

    mdphi = mult * phi - phiA * CUDA_DEG2RAD_F;
    sdphi = sinf(mdphi);
    *v    = cpA * (1.0F + cosf(mdphi));
    *f    = -cpA * mult * sdphi;
}

template<bool calcVir>
__device__ static void do_dih_fup_gpu(const int            i,
                                      const int            j,
                                      const int            k,
                                      const int            l,
                                      const float          ddphi,
                                      const float3         r_ij,
                                      const float3         r_kj,
                                      const float3         r_kl,
                                      const float3         m,
                                      const float3         n,
                                      float3               gm_f[],
                                      float3               sm_fShiftLoc[],
                                      const PbcAiuc&       pbcAiuc,
                                      const float4         gm_xq[],
                                      const int            t1,
                                      const int            t2,
                                      const int gmx_unused t3)
{
    float iprm  = norm2(m);
    float iprn  = norm2(n);
    float nrkj2 = norm2(r_kj);
    float toler = nrkj2 * GMX_REAL_EPS;
    if ((iprm > toler) && (iprn > toler))
    {
        float  nrkj_1 = rsqrtf(nrkj2); // replacing std::invsqrt call
        float  nrkj_2 = nrkj_1 * nrkj_1;
        float  nrkj   = nrkj2 * nrkj_1;
        float  a      = -ddphi * nrkj / iprm;
        float3 f_i    = a * m;
        float  b      = ddphi * nrkj / iprn;
        float3 f_l    = b * n;
        float  p      = iprod(r_ij, r_kj);
        p *= nrkj_2;
        float q = iprod(r_kl, r_kj);
        q *= nrkj_2;
        float3 uvec = p * f_i;
        float3 vvec = q * f_l;
        float3 svec = uvec - vvec;
        float3 f_j  = f_i - svec;
        float3 f_k  = f_l + svec;

        atomicAdd(&gm_f[i], f_i);
        atomicAdd(&gm_f[j], -f_j);
        atomicAdd(&gm_f[k], -f_k);
        atomicAdd(&gm_f[l], f_l);

        if (calcVir)
        {
            float3 dx_jl;
            int    t3 = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[l], gm_xq[j], dx_jl);

            atomicAdd(&sm_fShiftLoc[t1], f_i);
            atomicAdd(&sm_fShiftLoc[gmx::c_centralShiftIndex], -f_j);
            atomicAdd(&sm_fShiftLoc[t2], -f_k);
            atomicAdd(&sm_fShiftLoc[t3], f_l);
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void pdihs_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          const float4    gm_xq[],
                          float3          gm_f[],
                          float3          sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        float3 r_ij;
        float3 r_kj;
        float3 r_kl;
        float3 m;
        float3 n;
        int    t1;
        int    t2;
        int    t3;
        float  phi = dih_angle_gpu<calcVir>(
                gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc, &r_ij, &r_kj, &r_kl, &m, &n, &t1, &t2, &t3);

        float vpd;
        float ddphi;
        dopdihs_gpu(d_forceparams[type].pdihs.cpA,
                    d_forceparams[type].pdihs.phiA,
                    d_forceparams[type].pdihs.mult,
                    phi,
                    &vpd,
                    &ddphi);

        if (calcEner)
        {
            *vtot_loc += vpd;
        }

        do_dih_fup_gpu<calcVir>(
                ai, aj, ak, al, ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc, pbcAiuc, gm_xq, t1, t2, t3);
    }
}

template<bool calcVir, bool calcEner>
__device__ void rbdihs_gpu(const int       i,
                           float*          vtot_loc,
                           const int       numBonds,
                           const t_iatom   d_forceatoms[],
                           const t_iparams d_forceparams[],
                           const float4    gm_xq[],
                           float3          gm_f[],
                           float3          sm_fShiftLoc[],
                           const PbcAiuc   pbcAiuc)
{
    constexpr float c0 = 0.0F, c1 = 1.0F, c2 = 2.0F, c3 = 3.0F, c4 = 4.0F, c5 = 5.0F;

    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        float3 r_ij;
        float3 r_kj;
        float3 r_kl;
        float3 m;
        float3 n;
        int    t1;
        int    t2;
        int    t3;
        float  phi = dih_angle_gpu<calcVir>(
                gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc, &r_ij, &r_kj, &r_kl, &m, &n, &t1, &t2, &t3);

        /* Change to polymer convention */
        if (phi < c0)
        {
            phi += HIP_PI_F;
        }
        else
        {
            phi -= HIP_PI_F;
        }
        float cos_phi = cosf(phi);
        /* Beware of accuracy loss, cannot use 1-sqrt(cos^2) ! */
        float sin_phi = sinf(phi);

        float parm[NR_RBDIHS];
        for (int j = 0; j < NR_RBDIHS; j++)
        {
            parm[j] = d_forceparams[type].rbdihs.rbcA[j];
        }
        /* Calculate cosine powers */
        /* Calculate the energy */
        /* Calculate the derivative */
        float v      = parm[0];
        float ddphi  = c0;
        float cosfac = c1;

        float rbp = parm[1];
        ddphi += rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[2];
        ddphi += c2 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[3];
        ddphi += c3 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[4];
        ddphi += c4 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[5];
        ddphi += c5 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }

        ddphi = -ddphi * sin_phi;

        do_dih_fup_gpu<calcVir>(
                ai, aj, ak, al, ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc, pbcAiuc, gm_xq, t1, t2, t3);
        if (calcEner)
        {
            *vtot_loc += v;
        }
    }
}

__device__ __forceinline__ static void make_dp_periodic_gpu(float* dp)
{
    /* dp cannot be outside (-pi,pi) */
    if (*dp >= HIP_PI_F)
    {
        *dp -= 2.0F * HIP_PI_F;
    }
    else if (*dp < -HIP_PI_F)
    {
        *dp += 2.0F * HIP_PI_F;
    }
}

template<bool calcVir, bool calcEner>
__device__ void idihs_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          const float4    gm_xq[],
                          float3          gm_f[],
                          float3          sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        float3 r_ij;
        float3 r_kj;
        float3 r_kl;
        float3 m;
        float3 n;
        int    t1;
        int    t2;
        int    t3;
        float  phi = dih_angle_gpu<calcVir>(
                gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc, &r_ij, &r_kj, &r_kl, &m, &n, &t1, &t2, &t3);

        /* phi can jump if phi0 is close to Pi/-Pi, which will cause huge
         * force changes if we just apply a normal harmonic.
         * Instead, we first calculate phi-phi0 and take it modulo (-Pi,Pi).
         * This means we will never have the periodicity problem, unless
         * the dihedral is Pi away from phiO, which is very unlikely due to
         * the potential.
         */
        float kA = d_forceparams[type].harmonic.krA;
        float pA = d_forceparams[type].harmonic.rA;

        float phi0 = pA * CUDA_DEG2RAD_F;

        float dp = phi - phi0;

        make_dp_periodic_gpu(&dp);

        float ddphi = -kA * dp;

        do_dih_fup_gpu<calcVir>(
                ai, aj, ak, al, -ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc, pbcAiuc, gm_xq, t1, t2, t3);

        if (calcEner)
        {
            *vtot_loc += -0.5F * ddphi * dp;
        }
    }
}

/*! \brief Map indices into the periodic grid */
__device__ static int cmap_setup_grid_index(int ip, int grid_spacing, int* ipm1, int* ipp1, int* ipp2)
{
    int im1, ip1, ip2;

    if (ip < 0)
    {
        ip = ip + grid_spacing - 1;
    }
    else if (ip > grid_spacing)
    {
        ip = ip - grid_spacing - 1;
    }

    im1 = ip - 1;
    ip1 = ip + 1;
    ip2 = ip + 2;

    if (ip == 0)
    {
        im1 = grid_spacing - 1;
    }
    else if (ip == grid_spacing - 2)
    {
        ip2 = 0;
    }
    else if (ip == grid_spacing - 1)
    {
        ip1 = 0;
        ip2 = 1;
    }

    *ipm1 = im1;
    *ipp1 = ip1;
    *ipp2 = ip2;

    return ip;
}

__device__ static float3 processCmapForceComponent(const float a,
                                                   const float b,
                                                   const float df,
                                                   const float gaa,
                                                   const float fga,
                                                   const float gbb,
                                                   const float hgb)
{
    float3 result{ gaa * a, fga * a - hgb * b, gbb * b }; // mapping x <-> f, y <-> g, z <-> h
    return result * df;
}

__device__ static float4 applyCmapForceComponent(const float3 forceComponent)
{
    // forceComponent mapping is x <-> f, y <-> g, z <-> h
    float4 forces;
    forces.x = forceComponent.x;
    forces.y = -forceComponent.x - forceComponent.y;
    forces.z = forceComponent.z + forceComponent.y;
    forces.w = -forceComponent.z;
    return forces;
}

template<bool calcVir>
__device__ static void accumulateCmapForces(float3        gm_f[],
                                            const float4  gm_xq[],
                                            float3        sm_fShiftLoc[],
                                            const PbcAiuc pbcAiuc,
                                            float3        r_ij,
                                            float3        r_kj,
                                            float3        r_kl,
                                            float3        a,
                                            float3        b,
                                            float3        h,
                                            float         ra2r,
                                            float         rb2r,
                                            float         rgr,
                                            float         rg,
                                            int           ai,
                                            int           aj,
                                            int           ak,
                                            int           al,
                                            float         df,
                                            int           t1,
                                            int           t2)
{
    const float fg  = iprod(r_ij, r_kj);
    const float hg  = iprod(r_kl, r_kj);
    const float fga = fg * ra2r * rgr;
    const float hgb = hg * rb2r * rgr;
    const float gaa = -ra2r * rg;
    const float gbb = rb2r * rg;

    float3 f_i, f_j, f_k, f_l;
    float4 forceX = applyCmapForceComponent(processCmapForceComponent(a.x, b.x, df, gaa, fga, gbb, hgb));
    float4 forceY = applyCmapForceComponent(processCmapForceComponent(a.y, b.y, df, gaa, fga, gbb, hgb));
    float4 forceZ = applyCmapForceComponent(processCmapForceComponent(a.z, b.z, df, gaa, fga, gbb, hgb));
    f_i.x         = forceX.x;
    f_j.x         = forceX.y;
    f_k.x         = forceX.z;
    f_l.x         = forceX.w;

    f_i.y = forceY.x;
    f_j.y = forceY.y;
    f_k.y = forceY.z;
    f_l.y = forceY.w;

    f_i.z = forceZ.x;
    f_j.z = forceZ.y;
    f_k.z = forceZ.z;
    f_l.z = forceZ.w;

    atomicAdd(&gm_f[ai], f_i);
    atomicAdd(&gm_f[aj], f_j); /* - f[i] - g[i] */
    atomicAdd(&gm_f[ak], f_k); /* h[i] + g[i] */
    atomicAdd(&gm_f[al], f_l); /* - h[i] */

    /* Shift forces */
    if (calcVir)
    {
        int t3 = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[al], gm_xq[aj], h);
        atomicAdd(&sm_fShiftLoc[t1], f_i);
        atomicAdd(&sm_fShiftLoc[gmx::c_centralShiftIndex], f_j);
        atomicAdd(&sm_fShiftLoc[t2], f_k);
        atomicAdd(&sm_fShiftLoc[t3], f_l);
    }
}

/*! \brief
Magic lookup table for cmap indices

This maps the entries in the CMAP tables to the atoms in the
interactions
*/
__device__ static const int loopIndex[4][4] = { { 0, 4, 8, 12 },
                                                { 1, 5, 9, 13 },
                                                { 2, 6, 10, 14 },
                                                { 3, 7, 11, 15 } };

template<bool calcVir, bool calcEner>
__device__ void cmap_gpu(const int                  i,
                         float*                     vtot_loc,
                         const int                  numBonds,
                         const t_iatom              d_forceatoms[],
                         const t_iparams            d_forceparams[],
                         const int                  cmapGridSpacing,
                         const DeviceBuffer<float>& d_cmapData,
                         const int                  dc_cmapGridSize,
                         const float4               gm_xq[],
                         float3                     gm_f[],
                         float3                     sm_fShiftLoc[],
                         const PbcAiuc              pbcAiuc)
{
    if (i < numBonds)
    {
        /* Five atoms are involved in the two torsions */
        const int type = d_forceatoms[6 * i];
        const int ai   = d_forceatoms[6 * i + 1];
        const int aj   = d_forceatoms[6 * i + 2];
        const int ak   = d_forceatoms[6 * i + 3];
        const int al   = d_forceatoms[6 * i + 4];
        const int am   = d_forceatoms[6 * i + 5];

        /* Which CMAP type is this */
        const int    cmapA          = d_forceparams[type].cmap.cmapA;
        const int    cmapAGridIndex = dc_cmapGridSize * cmapA;
        const float* cmapd          = d_cmapData + cmapAGridIndex;

        int ip1m1, ip2m1, ip1p1, ip2p1, ip1p2, ip2p2;

        /* First torsion */
        const int a1i = ai;
        const int a1j = aj;
        const int a1k = ak;
        const int a1l = al;

        float3 r1_ij, r1_kj, r1_kl, r2_ij, r2_kj, r2_kl, m, n;
        float3 h1, h2;
        int    t11, t21, t31, t12, t22, t32;
        float  phi1 = dih_angle_gpu<calcVir>(
                gm_xq[a1i], gm_xq[a1j], gm_xq[a1k], gm_xq[a1l], pbcAiuc, &r1_ij, &r1_kj, &r1_kl, &m, &n, &t11, &t21, &t31);

        const float cos_phi1 = cos(phi1);

        float3 a1 = cprod(r1_ij, r1_kj);
        float3 b1 = cprod(r1_kl, r1_kj);

        const int ki = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[a1i], gm_xq[a1k], h1);

        const float ra21 = iprod(a1, a1);       /* 5 */
        const float rb21 = iprod(b1, b1);       /* 5 */
        const float rg21 = iprod(r1_kj, r1_kj); /* 5 */
        const float rg1  = sqrt(rg21);

        const float rgr1  = 1.0F / rg1;
        const float ra2r1 = 1.0F / ra21;
        const float rb2r1 = 1.0F / rb21;
        const float rabr1 = sqrt(ra2r1 * rb2r1);

        const float sin_phi1 = rg1 * rabr1 * iprod(a1, h1) * (-1.0F);

        if (cos_phi1 < -0.5F || cos_phi1 > 0.5F)
        {
            phi1 = asin(sin_phi1);

            if (cos_phi1 < 0.0F)
            {
                if (phi1 > 0.0F)
                {
                    phi1 = HIP_PI_F - phi1;
                }
                else
                {
                    phi1 = -HIP_PI_F - phi1;
                }
            }
        }
        else
        {
            phi1 = acos(cos_phi1);

            if (sin_phi1 < 0.0F)
            {
                phi1 = -phi1;
            }
        }

        float xphi1 = phi1 + HIP_PI_F; /* 1 */

        /* Second torsion */
        const int a2i = aj;
        const int a2j = ak;
        const int a2k = al;
        const int a2l = am;

        float phi2 = dih_angle_gpu<calcVir>(
                gm_xq[a2i], gm_xq[a2j], gm_xq[a2k], gm_xq[a2l], pbcAiuc, &r2_ij, &r2_kj, &r2_kl, &m, &n, &t12, &t22, &t32);

        float cos_phi2 = cos(phi2);

        float3 a2 = cprod(r2_ij, r2_kj);
        float3 b2 = cprod(r2_kl, r2_kj);

        pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[a2i], gm_xq[a2k], h2);

        const float ra22 = iprod(a2, a2);       /* 5 */
        const float rb22 = iprod(b2, b2);       /* 5 */
        const float rg22 = iprod(r2_kj, r2_kj); /* 5 */
        const float rg2  = sqrt(rg22);

        const float rgr2  = 1.0F / rg2;
        const float ra2r2 = 1.0F / ra22;
        const float rb2r2 = 1.0F / rb22;
        const float rabr2 = sqrt(ra2r2 * rb2r2);

        const float sin_phi2 = rg2 * rabr2 * iprod(a2, h2) * (-1.0F);

        if (cos_phi2 < -0.5F || cos_phi2 > 0.5F)
        {
            phi2 = asin(sin_phi2);

            if (cos_phi2 < 0.0F)
            {
                if (phi2 > 0.0F)
                {
                    phi2 = HIP_PI_F - phi2;
                }
                else
                {
                    phi2 = -HIP_PI_F - phi2;
                }
            }
        }
        else
        {
            phi2 = acos(cos_phi2);

            if (sin_phi2 < 0.0F)
            {
                phi2 = -phi2;
            }
        }

        float xphi2 = phi2 + HIP_PI_F; /* 1 */
        /* Range mangling */
        if (xphi1 < 0)
        {
            xphi1 = xphi1 + 2 * HIP_PI_F;
        }
        else if (xphi1 >= 2 * HIP_PI_F)
        {
            xphi1 = xphi1 - 2 * HIP_PI_F;
        }

        if (xphi2 < 0)
        {
            xphi2 = xphi2 + 2 * HIP_PI_F;
        }
        else if (xphi2 >= 2 * HIP_PI_F)
        {
            xphi2 = xphi2 - 2 * HIP_PI_F;
        }

        /* Number of grid points */
        float dx = 2 * HIP_PI_F / cmapGridSpacing;

        /* Where on the grid are we */
        int iphi1 = static_cast<int>(xphi1 / dx);
        int iphi2 = static_cast<int>(xphi2 / dx);

        iphi1 = cmap_setup_grid_index(iphi1, cmapGridSpacing, &ip1m1, &ip1p1, &ip1p2);
        iphi2 = cmap_setup_grid_index(iphi2, cmapGridSpacing, &ip2m1, &ip2p1, &ip2p2);

        const int pos1 = iphi1 * cmapGridSpacing + iphi2;
        const int pos2 = ip1p1 * cmapGridSpacing + iphi2;
        const int pos3 = ip1p1 * cmapGridSpacing + ip2p1;
        const int pos4 = iphi1 * cmapGridSpacing + ip2p1;

        float4 ty, ty1, ty2, ty12, tx[4];
        ty.x = cmapd[pos1 * 4];
        ty.y = cmapd[pos2 * 4];
        ty.z = cmapd[pos3 * 4];
        ty.w = cmapd[pos4 * 4];

        ty1.x = cmapd[pos1 * 4 + 1];
        ty1.y = cmapd[pos2 * 4 + 1];
        ty1.z = cmapd[pos3 * 4 + 1];
        ty1.w = cmapd[pos4 * 4 + 1];

        ty2.x = cmapd[pos1 * 4 + 2];
        ty2.y = cmapd[pos2 * 4 + 2];
        ty2.z = cmapd[pos3 * 4 + 2];
        ty2.w = cmapd[pos4 * 4 + 2];

        ty12.x = cmapd[pos1 * 4 + 3];
        ty12.y = cmapd[pos2 * 4 + 3];
        ty12.z = cmapd[pos3 * 4 + 3];
        ty12.w = cmapd[pos4 * 4 + 3];

        /* Switch to degrees */
        dx    = 360.0F / cmapGridSpacing;
        xphi1 = xphi1 * CUDA_RAD2DEG_F;
        xphi2 = xphi2 * CUDA_RAD2DEG_F;


        tx[0] = ty;
        tx[1] = ty1 * dx;
        tx[2] = ty2 * dx;
        tx[3] = ty12 * dx * dx;
        // for (int i = 0; i < 4; i++) /* 16 */
        // {
        //     tx[i]      = ty[i];
        //     tx[i + 4]  = ty1[i] * dx;
        //     tx[i + 8]  = ty2[i] * dx;
        //     tx[i + 12] = ty12[i] * dx * dx;
        // }

        float tc[16] = { 0.0F };
        for (int idx = 0; idx < 16; idx++) /* 1056 */
        {
            int kIndex = 0;
            for (int k = 0; k < 4; k++)
            {
                tc[idx] += cmap_coeff_matrix[kIndex++ * 16 + idx] * tx[k].x;
                tc[idx] += cmap_coeff_matrix[kIndex++ * 16 + idx] * tx[k].y;
                tc[idx] += cmap_coeff_matrix[kIndex++ * 16 + idx] * tx[k].z;
                tc[idx] += cmap_coeff_matrix[kIndex++ * 16 + idx] * tx[k].w;
            }
        }

        const float tt = (xphi1 - iphi1 * dx) / dx;
        const float tu = (xphi2 - iphi2 * dx) / dx;

        float e   = 0.0F;
        float df1 = 0.0F;
        float df2 = 0.0F;

        for (int i = 3; i >= 0; i--)
        {
            int l1 = loopIndex[i][3];
            int l2 = loopIndex[i][2];
            int l3 = loopIndex[i][1];

            e = tt * e + ((tc[i * 4 + 3] * tu + tc[i * 4 + 2]) * tu + tc[i * 4 + 1]) * tu + tc[i * 4];
            df1 = tu * df1 + (3.0F * tc[l1] * tt + 2.0 * tc[l2]) * tt + tc[l3];
            df2 = tt * df2 + (3.0F * tc[i * 4 + 3] * tu + 2.0F * tc[i * 4 + 2]) * tu + tc[i * 4 + 1];
        }

        const float fac = CUDA_RAD2DEG_F / dx;
        df1             = df1 * fac;
        df2             = df2 * fac;

        /* CMAP energy */
        if (calcEner)
        {
            *vtot_loc += e;
        }

        /* Do forces - first torsion */
        accumulateCmapForces<calcVir>(gm_f,
                                      gm_xq,
                                      sm_fShiftLoc,
                                      pbcAiuc,
                                      r1_ij,
                                      r1_kj,
                                      r1_kl,
                                      a1,
                                      b1,
                                      h1,
                                      ra2r1,
                                      rb2r1,
                                      rgr1,
                                      rg1,
                                      a1i,
                                      a1j,
                                      a1k,
                                      a1l,
                                      df1,
                                      t11,
                                      t21);

        /* Do forces - second torsion */
        accumulateCmapForces<calcVir>(gm_f,
                                      gm_xq,
                                      sm_fShiftLoc,
                                      pbcAiuc,
                                      r2_ij,
                                      r2_kj,
                                      r2_kl,
                                      a2,
                                      b2,
                                      h2,
                                      ra2r2,
                                      rb2r2,
                                      rgr2,
                                      rg2,
                                      a2i,
                                      a2j,
                                      a2k,
                                      a2l,
                                      df2,
                                      t12,
                                      t22);
    }
}

template<bool calcVir, bool calcEner>
__device__ void pairs_gpu(const int       i,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams iparams[],
                          const float4    gm_xq[],
                          float3          gm_f[],
                          float3          sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc,
                          const float     scale_factor,
                          float*          vtotVdw_loc,
                          float*          vtotElec_loc)
{
    if (i < numBonds)
    {
        // TODO this should be made into a separate type, the GPU and CPU sizes should be compared
        const int3 pairData = *(reinterpret_cast<const int3*>(d_forceatoms + 3 * i));
        int        type     = pairData.x;
        int        ai       = pairData.y;
        int        aj       = pairData.z;

        float qq  = gm_xq[ai].w * gm_xq[aj].w;
        float c6  = iparams[type].lj14.c6A;
        float c12 = iparams[type].lj14.c12A;

        /* Do we need to apply full periodic boundary conditions? */
        float3 dr;
        int    fshift_index = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[aj], dr);

        float r2    = norm2(dr);
        float rinv  = rsqrtf(r2);
        float rinv2 = rinv * rinv;
        float rinv6 = rinv2 * rinv2 * rinv2;

        /* Calculate the Coulomb force * r */
        float velec = scale_factor * qq * rinv;

        /* Calculate the LJ force * r and add it to the Coulomb part */
        float fr = (12.0F * c12 * rinv6 - 6.0F * c6) * rinv6 + velec;

        float  finvr = fr * rinv2;
        float3 f     = finvr * dr;

        /* Add the forces */
        atomicAdd(&gm_f[ai], f);
        atomicAdd(&gm_f[aj], -f);
        if (calcVir && fshift_index != gmx::c_centralShiftIndex)
        {
            atomicAdd(&sm_fShiftLoc[fshift_index], f);
            atomicAdd(&sm_fShiftLoc[gmx::c_centralShiftIndex], -f);
        }

        if (calcEner)
        {
            *vtotVdw_loc += (c12 * rinv6 - c6) * rinv6;
            *vtotElec_loc += velec;
        }
    }
}

namespace gmx
{

template<bool calcVir, bool calcEner>
__global__ void exec_kernel_gpu(BondedCudaKernelParameters kernelParams, float4* gm_xq, float3* gm_f, float3* gm_fShift)
{
    assert(blockDim.y == 1 && blockDim.z == 1);
    const int tid          = blockIdx.x * blockDim.x + threadIdx.x;
    float     vtot_loc     = 0;
    float     vtotVdw_loc  = 0;
    float     vtotElec_loc = 0;

    extern __shared__ char sm_dynamicShmem[];
    char*                  sm_nextSlotPtr = sm_dynamicShmem;
    float3*                sm_fShiftLoc   = reinterpret_cast<float3*>(sm_nextSlotPtr);
    sm_nextSlotPtr += c_numShiftVectors * sizeof(float3);

    if (calcVir)
    {
        if (threadIdx.x < c_numShiftVectors)
        {
            sm_fShiftLoc[threadIdx.x] = make_float3(0.0F, 0.0F, 0.0F);
        }
        __syncthreads();
    }

    int  fType;
    bool threadComputedPotential = false;
#pragma unroll
    for (int j = 0; j < numFTypesOnGpu; j++)
    {
        if (tid >= kernelParams.fTypeRangeStart[j] && tid <= kernelParams.fTypeRangeEnd[j])
        {
            const int      numBonds        = kernelParams.numFTypeBonds[j];
            int            fTypeTid        = tid - kernelParams.fTypeRangeStart[j];
            const t_iatom* iatoms          = kernelParams.d_iatoms[j];
            const auto     cmapData        = kernelParams.d_cmapData;
            const int      cmapGridSpacing = kernelParams.d_cmapGridSpacing;
            const int      cmapGridSize    = kernelParams.dc_cmapGridSize;
            fType                          = kernelParams.fTypesOnGpu[j];
            if (calcEner)
            {
                threadComputedPotential = true;
            }

            switch (fType)
            {
                case F_BONDS:
                    bonds_gpu<calcVir, calcEner>(fTypeTid,
                                                 &vtot_loc,
                                                 numBonds,
                                                 iatoms,
                                                 kernelParams.d_forceParams,
                                                 gm_xq,
                                                 gm_f,
                                                 sm_fShiftLoc,
                                                 kernelParams.pbcAiuc);
                    break;
                case F_ANGLES:
                    angles_gpu<calcVir, calcEner>(fTypeTid,
                                                  &vtot_loc,
                                                  numBonds,
                                                  iatoms,
                                                  kernelParams.d_forceParams,
                                                  gm_xq,
                                                  gm_f,
                                                  sm_fShiftLoc,
                                                  kernelParams.pbcAiuc);
                    break;
                case F_UREY_BRADLEY:
                    urey_bradley_gpu<calcVir, calcEner>(fTypeTid,
                                                        &vtot_loc,
                                                        numBonds,
                                                        iatoms,
                                                        kernelParams.d_forceParams,
                                                        gm_xq,
                                                        gm_f,
                                                        sm_fShiftLoc,
                                                        kernelParams.pbcAiuc);
                    break;
                case F_PDIHS:
                case F_PIDIHS:
                    pdihs_gpu<calcVir, calcEner>(fTypeTid,
                                                 &vtot_loc,
                                                 numBonds,
                                                 iatoms,
                                                 kernelParams.d_forceParams,
                                                 gm_xq,
                                                 gm_f,
                                                 sm_fShiftLoc,
                                                 kernelParams.pbcAiuc);
                    break;
                case F_RBDIHS:
                    rbdihs_gpu<calcVir, calcEner>(fTypeTid,
                                                  &vtot_loc,
                                                  numBonds,
                                                  iatoms,
                                                  kernelParams.d_forceParams,
                                                  gm_xq,
                                                  gm_f,
                                                  sm_fShiftLoc,
                                                  kernelParams.pbcAiuc);
                    break;
                case F_IDIHS:
                    idihs_gpu<calcVir, calcEner>(fTypeTid,
                                                 &vtot_loc,
                                                 numBonds,
                                                 iatoms,
                                                 kernelParams.d_forceParams,
                                                 gm_xq,
                                                 gm_f,
                                                 sm_fShiftLoc,
                                                 kernelParams.pbcAiuc);
                    break;
                case F_CMAP:
                    cmap_gpu<calcVir, calcEner>(fTypeTid,
                                                &vtot_loc,
                                                numBonds,
                                                iatoms,
                                                kernelParams.d_forceParams,
                                                cmapGridSpacing,
                                                cmapData,
                                                cmapGridSize,
                                                kernelParams.d_xq,
                                                kernelParams.d_f,
                                                sm_fShiftLoc,
                                                kernelParams.pbcAiuc);
                    break;
                case F_LJ14:
                    pairs_gpu<calcVir, calcEner>(fTypeTid,
                                                 numBonds,
                                                 iatoms,
                                                 kernelParams.d_forceParams,
                                                 gm_xq,
                                                 gm_f,
                                                 sm_fShiftLoc,
                                                 kernelParams.pbcAiuc,
                                                 kernelParams.electrostaticsScaleFactor,
                                                 &vtotVdw_loc,
                                                 &vtotElec_loc);
                    break;
            }
            break;
        }
    }

    if (threadComputedPotential)
    {
        float* vtotVdw  = kernelParams.d_vTot + F_LJ14;
        float* vtotElec = kernelParams.d_vTot + F_COUL14;

        // Stage atomic accumulation through shared memory:
        // each warp will accumulate its own partial sum
        // and then a single thread per warp will accumulate this to the global sum

        int numWarps = blockDim.x / warpSize;
        int warpId   = threadIdx.x / warpSize;

        // Shared memory variables to hold block-local partial sum
        float* sm_vTot = reinterpret_cast<float*>(sm_nextSlotPtr);
        sm_nextSlotPtr += numWarps * sizeof(float);
        float* sm_vTotVdw = reinterpret_cast<float*>(sm_nextSlotPtr);
        sm_nextSlotPtr += numWarps * sizeof(float);
        float* sm_vTotElec = reinterpret_cast<float*>(sm_nextSlotPtr);

        if (threadIdx.x % warpSize == 0)
        {
            // One thread per warp initializes to zero
            sm_vTot[warpId]     = 0.;
            sm_vTotVdw[warpId]  = 0.;
            sm_vTotElec[warpId] = 0.;
        }
        __syncwarp(); // All threads in warp must wait for initialization

        // Perform warp-local accumulation in shared memory
        atomicAdd(sm_vTot + warpId, vtot_loc);
        atomicAdd(sm_vTotVdw + warpId, vtotVdw_loc);
        atomicAdd(sm_vTotElec + warpId, vtotElec_loc);

        __syncwarp(); // Ensure all threads in warp have completed
        if (threadIdx.x % warpSize == 0)
        { // One thread per warp accumulates partial sum into global sum
            atomicAdd(kernelParams.d_vTot + fType, sm_vTot[warpId]);
            atomicAdd(vtotVdw, sm_vTotVdw[warpId]);
            atomicAdd(vtotElec, sm_vTotElec[warpId]);
        }
    }
    /* Accumulate shift vectors from shared memory to global memory on the first c_numShiftVectors threads of the block. */
    if (calcVir)
    {
        __syncthreads();
        if (threadIdx.x < c_numShiftVectors)
        {
            atomicAdd(gm_fShift[threadIdx.x], sm_fShiftLoc[threadIdx.x]);
        }
    }
}


/*-------------------------------- End CUDA kernels-----------------------------*/


template<bool calcVir, bool calcEner>
void ListedForcesGpu::Impl::launchKernel()
{
    GMX_ASSERT(haveInteractions_,
               "Cannot launch bonded GPU kernels unless bonded GPU work was scheduled");

    wallcycle_start_nocount(wcycle_, WallCycleCounter::LaunchGpu);
    wallcycle_sub_start(wcycle_, WallCycleSubCounter::LaunchGpuBonded);

    int fTypeRangeEnd = kernelParams_.fTypeRangeEnd[numFTypesOnGpu - 1];

    if (fTypeRangeEnd < 0)
    {
        return;
    }

    auto kernelPtr = exec_kernel_gpu<calcVir, calcEner>;

    const auto kernelArgs = prepareGpuKernelArguments(
            kernelPtr, kernelLaunchConfig_, &kernelParams_, &d_xq_, &d_f_, &d_fShift_);

    launchGpuKernel(kernelPtr,
                    kernelLaunchConfig_,
                    deviceStream_,
                    nullptr,
                    "exec_kernel_gpu<calcVir, calcEner>",
                    kernelArgs);

    wallcycle_sub_stop(wcycle_, WallCycleSubCounter::LaunchGpuBonded);
    wallcycle_stop(wcycle_, WallCycleCounter::LaunchGpu);
}

void ListedForcesGpu::launchKernel(const gmx::StepWorkload& stepWork)
{
    if (stepWork.computeEnergy)
    {
        // When we need the energy, we also need the virial
        impl_->launchKernel<true, true>();
    }
    else if (stepWork.computeVirial)
    {
        impl_->launchKernel<true, false>();
    }
    else
    {
        impl_->launchKernel<false, false>();
    }
}

} // namespace gmx
